#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "convolutional_layer.h"
#include "gemm.h"
#include "blas.h"
#include "im2col.h"
#include "col2im.h"
#include "utils.h"
#include "hip/hip_runtime.h"
}

__global__ void binarize_filters_kernel(float *filters, int n, int size, float *binary)
{
    int f = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (f >= n) return;
    int i = 0;
    float mean = 0;
    for(i = 0; i < size; ++i){
        mean += abs(filters[f*size + i]);
    }
    mean = mean / size;
    for(i = 0; i < size; ++i){
        binary[f*size + i] = (filters[f*size + i] > 0) ? mean : -mean;
    }
}

__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    if(offset < size) output[(batch*n+filter)*size + offset] *= biases[filter];
}

void scale_bias_gpu(float *output, float *biases, int batch, int n, int size)
{
    dim3 dimGrid((size-1)/BLOCK + 1, n, batch);
    dim3 dimBlock(BLOCK, 1, 1);

    scale_bias_kernel<<<dimGrid, dimBlock>>>(output, biases, n, size);
    check_error(hipPeekAtLastError());
}

__global__ void backward_scale_kernel(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates)
{
    __shared__ float part[BLOCK];
    int i,b;
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;
    for(b = 0; b < batch; ++b){
        for(i = 0; i < size; i += BLOCK){
            int index = p + i + size*(filter + n*b);
            sum += (p+i < size) ? delta[index]*x_norm[index] : 0;
        }
    }
    part[p] = sum;
    __syncthreads();
    if (p == 0) {
        for(i = 0; i < BLOCK; ++i) scale_updates[filter] += part[i];
    }
}

void binarize_filters_gpu(float *filters, int n, int size, float *mean)
{
    binarize_filters_kernel<<<cuda_gridsize(n), BLOCK>>>(filters, n, size, mean);
    check_error(hipPeekAtLastError());
}

void backward_scale_gpu(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates)
{
    backward_scale_kernel<<<n, BLOCK>>>(x_norm, delta, batch, n, size, scale_updates);
    check_error(hipPeekAtLastError());
}

__global__ void add_bias_kernel(float *output, float *biases, int n, int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    if(offset < size) output[(batch*n+filter)*size + offset] += biases[filter];
}

void add_bias_gpu(float *output, float *biases, int batch, int n, int size)
{
    dim3 dimGrid((size-1)/BLOCK + 1, n, batch);
    dim3 dimBlock(BLOCK, 1, 1);

    add_bias_kernel<<<dimGrid, dimBlock>>>(output, biases, n, size);
    check_error(hipPeekAtLastError());
}

__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size)
{
    __shared__ float part[BLOCK];
    int i,b;
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;
    for(b = 0; b < batch; ++b){
        for(i = 0; i < size; i += BLOCK){
            int index = p + i + size*(filter + n*b);
            sum += (p+i < size) ? delta[index] : 0;
        }
    }
    part[p] = sum;
    __syncthreads();
    if (p == 0) {
        for(i = 0; i < BLOCK; ++i) bias_updates[filter] += part[i];
    }
}

void backward_bias_gpu(float *bias_updates, float *delta, int batch, int n, int size)
{
    backward_bias_kernel<<<n, BLOCK>>>(bias_updates, delta, batch, n, size);
    check_error(hipPeekAtLastError());
}

void swap_binary(convolutional_layer l)
{
        float *swap = l.filters_gpu;
        l.filters_gpu = l.binary_filters_gpu;
        l.binary_filters_gpu = swap;
}

void forward_convolutional_layer_gpu(convolutional_layer l, network_state state)
{
    int i;
    int m = l.n;
    int k = l.size*l.size*l.c;
    int n = convolutional_out_height(l)*
        convolutional_out_width(l);

    fill_ongpu(l.outputs*l.batch, 0, l.output_gpu, 1);
    if(l.binary){
        binarize_filters_gpu(l.filters_gpu, l.n, l.c*l.size*l.size, l.binary_filters_gpu);
        swap_binary(l);
    }

    for(i = 0; i < l.batch; ++i){
        im2col_ongpu(state.input + i*l.c*l.h*l.w, l.c,  l.h,  l.w,  l.size,  l.stride, l.pad, l.col_image_gpu);
        float * a = l.filters_gpu;
        float * b = l.col_image_gpu;
        float * c = l.output_gpu;
        gemm_ongpu(0,0,m,n,k,1.,a,k,b,n,1.,c+i*m*n,n);
    }

    if(l.batch_normalize){
        if(state.train){
            fast_mean_gpu(l.output_gpu, l.batch, l.n, l.out_h*l.out_w, l.mean_gpu);
            fast_variance_gpu(l.output_gpu, l.mean_gpu, l.batch, l.n, l.out_h*l.out_w, l.variance_gpu);

            scal_ongpu(l.n, .95, l.rolling_mean_gpu, 1);
            axpy_ongpu(l.n, .05, l.mean_gpu, 1, l.rolling_mean_gpu, 1);
            scal_ongpu(l.n, .95, l.rolling_variance_gpu, 1);
            axpy_ongpu(l.n, .05, l.variance_gpu, 1, l.rolling_variance_gpu, 1);

            copy_ongpu(l.outputs*l.batch, l.output_gpu, 1, l.x_gpu, 1);
            normalize_gpu(l.output_gpu, l.mean_gpu, l.variance_gpu, l.batch, l.n, l.out_h*l.out_w);
            copy_ongpu(l.outputs*l.batch, l.output_gpu, 1, l.x_norm_gpu, 1);
        } else {
            normalize_gpu(l.output_gpu, l.rolling_mean_gpu, l.rolling_variance_gpu, l.batch, l.n, l.out_h*l.out_w);
        }

        scale_bias_gpu(l.output_gpu, l.scales_gpu, l.batch, l.n, l.out_h*l.out_w);
    }
    add_bias_gpu(l.output_gpu, l.biases_gpu, l.batch, l.n, n);

    activate_array_ongpu(l.output_gpu, m*n*l.batch, l.activation);
    if(l.binary) swap_binary(l);
}

void backward_convolutional_layer_gpu(convolutional_layer l, network_state state)
{
    int i;
    int m = l.n;
    int n = l.size*l.size*l.c;
    int k = convolutional_out_height(l)*
        convolutional_out_width(l);

    gradient_array_ongpu(l.output_gpu, m*k*l.batch, l.activation, l.delta_gpu);

    backward_bias_gpu(l.bias_updates_gpu, l.delta_gpu, l.batch, l.n, k);

    if(l.batch_normalize){
        backward_scale_gpu(l.x_norm_gpu, l.delta_gpu, l.batch, l.n, l.out_w*l.out_h, l.scale_updates_gpu);

        scale_bias_gpu(l.delta_gpu, l.scales_gpu, l.batch, l.n, l.out_h*l.out_w);

        fast_mean_delta_gpu(l.delta_gpu, l.variance_gpu, l.batch, l.n, l.out_w*l.out_h, l.mean_delta_gpu);
        fast_variance_delta_gpu(l.x_gpu, l.delta_gpu, l.mean_gpu, l.variance_gpu, l.batch, l.n, l.out_w*l.out_h, l.variance_delta_gpu);
        normalize_delta_gpu(l.x_gpu, l.mean_gpu, l.variance_gpu, l.mean_delta_gpu, l.variance_delta_gpu, l.batch, l.n, l.out_w*l.out_h, l.delta_gpu);
    }

    for(i = 0; i < l.batch; ++i){
        float * a = l.delta_gpu;
        float * b = l.col_image_gpu;
        float * c = l.filter_updates_gpu;

        im2col_ongpu(state.input + i*l.c*l.h*l.w, l.c,  l.h,  l.w,  l.size,  l.stride, l.pad, l.col_image_gpu);
        gemm_ongpu(0,1,m,n,k,1,a + i*m*k,k,b,k,1,c,n);

        if(state.delta){
            if(l.binary) swap_binary(l);
            float * a = l.filters_gpu;
            float * b = l.delta_gpu;
            float * c = l.col_image_gpu;

            gemm_ongpu(1,0,n,k,m,1,a,n,b + i*k*m,k,0,c,k);

            col2im_ongpu(l.col_image_gpu, l.c,  l.h,  l.w,  l.size,  l.stride, l.pad, state.delta + i*l.c*l.h*l.w);
            if(l.binary) swap_binary(l);
        }
    }
}

void pull_convolutional_layer(convolutional_layer layer)
{
    cuda_pull_array(layer.filters_gpu, layer.filters, layer.c*layer.n*layer.size*layer.size);
    cuda_pull_array(layer.biases_gpu, layer.biases, layer.n);
    cuda_pull_array(layer.filter_updates_gpu, layer.filter_updates, layer.c*layer.n*layer.size*layer.size);
    cuda_pull_array(layer.bias_updates_gpu, layer.bias_updates, layer.n);
    if (layer.batch_normalize){
        cuda_pull_array(layer.scales_gpu, layer.scales, layer.n);
        cuda_pull_array(layer.rolling_mean_gpu, layer.rolling_mean, layer.n);
        cuda_pull_array(layer.rolling_variance_gpu, layer.rolling_variance, layer.n);
    }
}

void push_convolutional_layer(convolutional_layer layer)
{
    cuda_push_array(layer.filters_gpu, layer.filters, layer.c*layer.n*layer.size*layer.size);
    cuda_push_array(layer.biases_gpu, layer.biases, layer.n);
    cuda_push_array(layer.filter_updates_gpu, layer.filter_updates, layer.c*layer.n*layer.size*layer.size);
    cuda_push_array(layer.bias_updates_gpu, layer.bias_updates, layer.n);
    if (layer.batch_normalize){
        cuda_push_array(layer.scales_gpu, layer.scales, layer.n);
        cuda_push_array(layer.rolling_mean_gpu, layer.rolling_mean, layer.n);
        cuda_push_array(layer.rolling_variance_gpu, layer.rolling_variance, layer.n);
    }
}

void update_convolutional_layer_gpu(convolutional_layer layer, int batch, float learning_rate, float momentum, float decay)
{
    int size = layer.size*layer.size*layer.c*layer.n;

    axpy_ongpu(layer.n, learning_rate/batch, layer.bias_updates_gpu, 1, layer.biases_gpu, 1);
    scal_ongpu(layer.n, momentum, layer.bias_updates_gpu, 1);

    axpy_ongpu(layer.n, learning_rate/batch, layer.scale_updates_gpu, 1, layer.scales_gpu, 1);
    scal_ongpu(layer.n, momentum, layer.scale_updates_gpu, 1);

    axpy_ongpu(size, -decay*batch, layer.filters_gpu, 1, layer.filter_updates_gpu, 1);
    axpy_ongpu(size, learning_rate/batch, layer.filter_updates_gpu, 1, layer.filters_gpu, 1);
    scal_ongpu(size, momentum, layer.filter_updates_gpu, 1);
}


