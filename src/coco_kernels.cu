#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "network.h"
#include "detection_layer.h"
#include "cost_layer.h"
#include "utils.h"
#include "parser.h"
#include "box.h"
#include "image.h"
#include <sys/time.h>
}

#ifdef OPENCV
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/imgproc/imgproc.hpp"
extern "C" image ipl_to_image(IplImage* src);
extern "C" void convert_coco_detections(float *predictions, int classes, int num, int square, int side, int w, int h, float thresh, float **probs, box *boxes, int only_objectness);

extern "C" char *coco_classes[];
extern "C" image coco_labels[];

static float **probs;
static box *boxes;
static network net;
static image in   ;
static image in_s ;
static image det  ;
static image det_s;
static image disp ;
static cv::VideoCapture cap;
static float fps = 0;
static float demo_thresh = 0;

static const int frames = 3;
static float *predictions[frames];
static int demo_index = 0;
static image images[frames];
static float *avg;

void *fetch_in_thread_coco(void *ptr)
{
    cv::Mat frame_m;
    cap >> frame_m;
    IplImage frame = frame_m;
    in = ipl_to_image(&frame);
    rgbgr_image(in);
    in_s = resize_image(in, net.w, net.h);
    return 0;
}

void *detect_in_thread_coco(void *ptr)
{
    float nms = .4;

    detection_layer l = net.layers[net.n-1];
    float *X = det_s.data;
    float *prediction = network_predict(net, X);

    memcpy(predictions[demo_index], prediction, l.outputs*sizeof(float));
    mean_arrays(predictions, frames, l.outputs, avg);

    free_image(det_s);
    convert_coco_detections(avg, l.classes, l.n, l.sqrt, l.side, 1, 1, demo_thresh, probs, boxes, 0);
    if (nms > 0) do_nms(boxes, probs, l.side*l.side*l.n, l.classes, nms);
    printf("\033[2J");
    printf("\033[1;1H");
    printf("\nFPS:%.0f\n",fps);
    printf("Objects:\n\n");

    images[demo_index] = det;
    det = images[(demo_index + frames/2 + 1)%frames];
    demo_index = (demo_index + 1)%frames;

    draw_detections(det, l.side*l.side*l.n, demo_thresh, boxes, probs, coco_classes, coco_labels, 80);
    return 0;
}

extern "C" void demo_coco(char *cfgfile, char *weightfile, float thresh, int cam_index, const char *filename)
{
    demo_thresh = thresh;
    printf("YOLO demo\n");
    net = parse_network_cfg(cfgfile);
    if(weightfile){
        load_weights(&net, weightfile);
    }
    set_batch_network(&net, 1);

    srand(2222222);

    if(filename){
        cap.open(filename);
    }else{
        cap.open(cam_index);
    }

    if(!cap.isOpened()) error("Couldn't connect to webcam.\n");

    detection_layer l = net.layers[net.n-1];
    int j;

    avg = (float *) calloc(l.outputs, sizeof(float));
    for(j = 0; j < frames; ++j) predictions[j] = (float *) calloc(l.outputs, sizeof(float));
    for(j = 0; j < frames; ++j) images[j] = make_image(1,1,3);

    boxes = (box *)calloc(l.side*l.side*l.n, sizeof(box));
    probs = (float **)calloc(l.side*l.side*l.n, sizeof(float *));
    for(j = 0; j < l.side*l.side*l.n; ++j) probs[j] = (float *)calloc(l.classes, sizeof(float *));

    pthread_t fetch_thread;
    pthread_t detect_thread;

    fetch_in_thread_coco(0);
    det = in;
    det_s = in_s;

    fetch_in_thread_coco(0);
    detect_in_thread_coco(0);
    disp = det;
    det = in;
    det_s = in_s;

    while(1){
        struct timeval tval_before, tval_after, tval_result;
        gettimeofday(&tval_before, NULL);
        if(pthread_create(&fetch_thread, 0, fetch_in_thread_coco, 0)) error("Thread creation failed");
        if(pthread_create(&detect_thread, 0, detect_in_thread_coco, 0)) error("Thread creation failed");
        show_image(disp, "YOLO");
        free_image(disp);
        cvWaitKey(1);
        pthread_join(fetch_thread, 0);
        pthread_join(detect_thread, 0);

        disp  = det;
        det   = in;
        det_s = in_s;

        gettimeofday(&tval_after, NULL);
        timersub(&tval_after, &tval_before, &tval_result);
        float curr = 1000000.f/((long int)tval_result.tv_usec);
        fps = .9*fps + .1*curr;
    }
}
#else
extern "C" void demo_coco(char *cfgfile, char *weightfile, float thresh, int cam_index){
    fprintf(stderr, "YOLO-COCO demo needs OpenCV for webcam images.\n");
}
#endif

